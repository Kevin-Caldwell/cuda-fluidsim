#include "hip/hip_runtime.h"
/** Imports */
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// #include <thread>
#include <random>
#include <unistd.h>
// #include <limits.h>

#include "backups.h"
#include "config_reader.h"
#include "fsim_manager.cuh"
// #include "include/ppm_handler.h"
#include "array_utils.h"
#include "device_utils.h"
#include "img_thread.h"
#include "log.h"
#include "ppm_handler.h"

char wbuf[40];

#define DEBUG

void velocity_field_init(int elem_count, int buffer_size, int dim_x, int dim_y,
                         float *h_buffer, SimParams *params, float *d_u,
                         float *d_v, float *d_pressure) {
  for (int i = 0; i < elem_count; i++) {
    h_buffer[i] = params->offset_vel_x;
  }

  h_buffer[elem(dim_x / 2, dim_y / 2, dim_x)] = 0.5;

  // for (int i = dim_y / 3; i < 2 * dim_y / 3; i++)
  // {
  //     h_buffer[elem(dim_x / 2, i, dim_x)] = 0.1;
  // }

  // fill_random(h_buffer, elem_count);

  hipMemcpy(d_u, h_buffer, buffer_size, hipMemcpyHostToDevice);

  for (int i = 0; i < elem_count; i++) {
    h_buffer[i] = params->offset_vel_y;
  }

  hipMemcpy(d_v, h_buffer, buffer_size, hipMemcpyHostToDevice);

  for (int i = 0; i < elem_count; i++) {
    h_buffer[i] = 0.0;
  }

  hipMemcpy(d_pressure, h_buffer, buffer_size, hipMemcpyHostToDevice);
}

// char *h_chbuffer = NULL;

// void render_scalar_field(ppm_handler img_creator, int index,
//                          const char *annotation, int elem_count, float
//                          *d_data, float *h_buffer) {

//   char filename_buf[100];

//   hipMemcpy(h_buffer, d_data, elem_count * sizeof(float),
//              hipMemcpyDeviceToHost);
//   float max = utils::array::max_arr(h_buffer, elem_count);
//   float min = utils::array::min_arr(h_buffer, elem_count);
//   // printf("(%03f, %03f)", max, min);
//   utils::array::fl_to_char_arr(h_buffer, h_chbuffer, elem_count,
//                                255 / (max - min), min);
//   snprintf(filename_buf, 100, "temp/%s_%03d.ppm", annotation, index);
//   img_creator.write_ppm(filename_buf, h_chbuffer);
// }

void print_field(float *arr, int size_x, int size_y) {
  for (int i = 0; i < size_y; i++) {
    printf("[%f ", arr[size_x * i]);
    for (int j = 1; j < size_x; j++) {
      printf(", %f", arr[size_x * i + j]);
    }
    printf("]\n");
  }
  printf("\n");
}

int main(void) {
  if (gpu::check_cuda_dev()) {
    return -1;
  }

  long int time;
  utils::log::tick();

  SimParams params;
  SimParams *d_params = NULL;
  SimData h_data;
  SimData *d_data = NULL;

  float *d_pressure = NULL, *d_u = NULL, *d_v = NULL;
  float *d_temp0 = NULL, *d_temp1 = NULL;
  float *h_buffer = NULL;

  utils::config_reader::parse_fsim_config("fsim.config", &params);

  const int dim_x = params.dim_x, dim_y = params.dim_y;
  const float tf = params.tf;

  params.dx = params.size_x / (float)params.dim_x;
  params.dy = params.size_y / (float)params.dim_y;

  hipMalloc(&d_params, sizeof(SimParams));
  hipMemcpy(d_params, &params, sizeof(SimParams), hipMemcpyHostToDevice);

  int buffer_size = params.dim_x * params.dim_y * sizeof(float);
  int elem_count = params.dim_x * params.dim_y;

  // Allocate Memory for Velocity and Pressure Fields
  hipMalloc((void **)&d_pressure, buffer_size);
  hipMalloc((void **)&d_u, buffer_size);
  hipMalloc((void **)&d_v, buffer_size);
  // Allocate Fields for holding Intermediate Values
  hipMalloc((void **)&d_temp0, buffer_size);
  hipMalloc((void **)&d_temp1, buffer_size);

  // Allocate Temporary RAM Buffer to hold data to set/use Device Memory
  h_buffer = (float *)malloc(buffer_size);
  // h_chbuffer = (char *)malloc(elem_count * sizeof(char));

  dim3 blocks = {(unsigned)params.dim_x, (unsigned)params.dim_y};

  // Initialize SimData Struct for the host
  h_data = {.params = d_params,
            .u = d_u,
            .v = d_v,
            .pressure = d_pressure,
            .temp_0 = d_temp0,
            .temp_1 = d_temp1};

  hipMalloc((void **)&d_data, sizeof(SimData));
  hipMemcpy(d_data, &h_data, sizeof(SimData), hipMemcpyHostToDevice);

  printf("Successfully Allocated Memory...\n");

  utils::backup::setup_backup();

  velocity_field_init(elem_count, buffer_size, dim_x, dim_y, h_buffer, &params,
                      d_u, d_v, d_pressure);

  system("mkdir temp");

  bool save_arena = true;
  unsigned int iterations = 0;

  ppm_handler img_creater = ppm_handler(dim_x, dim_y, 0);

  time = utils::log::tock();
  printf("Setup Time: %ld ms ", time / 1000000);

  for (; params.t < tf; params.t += params.dt) {
    write(STDOUT_FILENO, "\r", 1);
    printf("%08f/%08f\t", params.t, tf, utils::log::tock() / 1000000);
    // save_arena = iterations % 1 == 0;

    // Copy U, V, P from GPU to Memory and save to csv for Debugging
    if (save_arena) {
      // printf("Render Time: %d\t", utils::log::tock() / 1000000);
      // render_scalar_field(img_creater, iterations, "u", elem_count, d_u,
      //                     h_buffer);
      // printf("%d\t", utils::log::tock() / 1000000);
      // render_scalar_field(img_creater, iterations, "v", elem_count, d_v,
      //                     h_buffer);
      // printf("%d\t", utils::log::tock() / 1000000);
      // render_scalar_field(img_creater, iterations, "pressure", elem_count,
      //                     d_pressure, h_buffer);
      // printf("%d\t", utils::log::tock() / 1000000);
    }

    // Iteratively Smooth Pressure
    for (int i = 0; i < params.smoothing; i++) {
      fluidsim::fsim_smooth_pressure(d_data, blocks);
      hipMemcpy(d_pressure, d_temp0, buffer_size, hipMemcpyDeviceToDevice);
    }
    printf("%d\t", utils::log::tock() / 1000000);

    // Update Velocity Vector Field
    fluidsim::fsim_update_u(d_data, blocks);
    fluidsim::fsim_update_v(d_data, blocks);
    printf("%d\t", utils::log::tock() / 1000000);

    hipMemcpy(d_u, d_temp0, buffer_size, hipMemcpyDeviceToDevice);
    hipMemcpy(d_v, d_temp1, buffer_size, hipMemcpyDeviceToDevice);
    printf("%d\t", utils::log::tock() / 1000000);

    fflush(0);
    iterations++;
  }

  printf("Freeing all Allocated Memory.\n");

  // Free all allocated Buffers
  free(h_buffer);
  hipFree(d_pressure);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_temp0);
  hipFree(d_temp1);

  system("./../../../vidgen.sh");
  system("rm -r temp");

  utils::backup::exit_backup(&params);
  return 0;
}